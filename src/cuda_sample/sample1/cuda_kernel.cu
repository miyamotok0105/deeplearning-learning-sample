#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

#define CUDA_SAFE_CALL(func) { \
    hipError_t err = (func); \
    if (err != hipSuccess) { \
        fprintf(stderr, "error [%d] : %s\n", err, hipGetErrorString(err)); \
        exit(err); \
    } \
}

// __global__関数はホストから呼び出せるデバイス側関数
// 戻り値は返せない
__global__ void addKernel(int *c, const int *a, const int *b, const int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) {
        //printf(">> blockIdx: %d  threadIdx: %d\n", blockIdx.x, threadIdx.x);
        return;
    }
    c[i] = a[i] + b[i];
}

void add(int *c, const int *a, const int *b, unsigned int n)
{
    int *dev_a;
    int *dev_b;
    int *dev_c;
    hipSetDevice(0);

    // 3つの配列領域(2入力/1出力)をGPU側に確保
    CUDA_SAFE_CALL( hipMalloc((void **)&dev_c, sizeof(int) * n) );
    CUDA_SAFE_CALL( hipMalloc((void **)&dev_a, sizeof(int) * n) );
    CUDA_SAFE_CALL( hipMalloc((void **)&dev_b, sizeof(int) * n) );
    // 2つの入力データ(配列)をホストからデバイスへ転送
    CUDA_SAFE_CALL( hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemcpy(dev_b, b, n * sizeof(int), hipMemcpyHostToDevice) );

    // カーネル呼び出し
    // Grid > Block > Thread
    // func_gpu << Dg, Db [ , Ns, S ] >>> (a, b, c);
    // Dg : グリッドサイズ (グリッド内のブロック数)
    // Db : ブロックサイズ (ブロック内のスレッド数)
    // Ns : シェアードメモリのサイズ. 省略時は 0
    // S  : ストリーム番号
    int th = 1024;
    int bl = (n / 1024) + 1;
    dim3 blocks(bl, 1, 1);
    dim3 threads(th, 1, 1);
    addKernel<<<blocks, threads>>>(dev_c, dev_a, dev_b, n);

    // カーネルの終了を待つ
    hipDeviceSynchronize();
    // 結果(配列)をデバイスからホストへ転送
    hipMemcpy(c, dev_c, n * sizeof(int), hipMemcpyDeviceToHost);
    // デバイス側メモリ開放
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    return;
}

#ifdef __cplusplus
extern "C" {
#endif
void cuda_kernel_exec(int n)
{
    int i;
    int *a = (int *)malloc(sizeof(int) * n);
    int *b = (int *)malloc(sizeof(int) * n);
    int *c = (int *)malloc(sizeof(int) * n);

    for (i=0; i<n; i++) {
      a[i] = i+1;
      b[i] = i-1;
      c[i] = 0;
    }
    // Add vectors in parallel.
    add(c, a, b, n);
    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipDeviceReset();
    free(a);
    free(b);
    free(c);
    return;
}
#ifdef __cplusplus
};
#endif
